#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <algorithm>
#include <cassert>
#include <cstring>
#include <iostream>
#include <unordered_map>
#include "constants.h"
#include "from_ivta.h"
using namespace std;

__global__ void calcPositions(long n, Sample *samples) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    Sample sample = samples[i];
    sample.v.x = sample.v.x * a_factor + sample.f.x * b_factor;
    sample.v.y = sample.v.y * a_factor + sample.f.y * b_factor;
    sample.f = {0, 0};
    sample.pos.x += sample.v.x;
    sample.pos.y += sample.v.y;
    samples[i] = sample;
  }
  return;
}

__global__ void calcForceComponents(int compNumber, DistElem *distances, Sample *samples) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < compNumber) {
    DistElem distance = distances[i];

    float2 posI = samples[distance.i].pos;
    float2 posJ = samples[distance.j].pos;

    float2 rv = posI;
    rv.x -= posJ.x;
    rv.y -= posJ.y;

    float r = sqrtf((posI.x - posJ.x) * (posI.x - posJ.x) +
                    (posI.y - posJ.y) * (posI.y - posJ.y));
    float D = distance.r;

    float energy = (r - D) / r;
    rv.x *= -energy;
    rv.y *= -energy;

    // distances are sorted by their type
    if (distance.type == etRandom) {
      rv.x *= w_random;
      rv.y *= w_random;
    }
    *distance.comp1 = rv;
    *distance.comp2 = {-rv.x, -rv.y};
  }
  return;
}

__global__ void applyForces(int n, Sample *samples) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    Sample sample = samples[i];

    for (int j = 0; j < sample.num_components; j++) {
      sample.f.x += sample.components[j].x;
      sample.f.y += sample.components[j].y;
    }

    samples[i] = sample;
  }
  return;
}

  // initialize pos in Samples
  // initialize num_components
__global__ void initializeSamples(int n, Sample *samples, float2 *positions, short *sampleFreq) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    Sample sample;
    sample.pos = positions[i];
    sample.v = sample.f = {0, 0};
    sample.num_components = sampleFreq[i];
    // FIXME - malloc can return NULL
    sample.components = (float2 *)malloc(sample.num_components * sizeof(float2));
    samples[i] = sample; 
  }
}

__global__ void initializeDistances(int nDst, DistElem *distances, short2 *dstIndexes, Sample *samples) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i<nDst) {
    DistElem dst = distances[i];
    dst.comp1 = &samples[dst.i].components[dstIndexes[i].x];
    dst.comp2 = &samples[dst.j].components[dstIndexes[i].y];
    distances[i] = dst;
  }
}

void IVHD::initializeHelperVectors() {
  /*
   * calculate number of distances for each sample 
   * and index of each distance for a given sample
   */
  short sampleFreq[positions.size()];
  for (int i=0; i<positions.size(); i++) {
    sampleFreq[i] = 0;
  }

  short2 dstIndexes[distances.size()];

  for (int i=0; i<distances.size(); i++) {
    dstIndexes[i] = {sampleFreq[distances[i].i]++, sampleFreq[distances[i].j]++};
  }

  // initialize samples
  short *d_sample_freq;
  cuCall(hipMalloc(&d_sample_freq, positions.size() * sizeof(short)));
  cuCall(hipMemcpy(d_sample_freq, sampleFreq,
                    sizeof(short) * positions.size(),
                    hipMemcpyHostToDevice));

  initializeSamples<<<positions.size() / 256 + 1, 256>>>(positions.size(), d_samples, d_positions, d_sample_freq);
  cuCall(hipFree(d_sample_freq));

  // initialize comps in Distances in device memory
  short2 *d_dst_indexes;
  cuCall(hipMalloc(&d_dst_indexes, distances.size() * sizeof(short2)));
  cuCall(hipMemcpy(d_dst_indexes, dstIndexes,
                    sizeof(short2) * distances.size(),
                    hipMemcpyHostToDevice));

  initializeDistances<<<distances.size() / 256 + 1, 256>>>(distances.size(), d_distances, d_dst_indexes, d_samples);
  cuCall(hipFree(d_dst_indexes));

  // optionally sort samples by number of their distances and
  // distances by i.e. dist.i to utilize cache better

  // generate sorted (by the smallest number of distances) list of sample
  // indexes
  /*vector<int> sampleIndexes;*/
  /*for (int i = 0; i < positions.size(); i++) {*/
  /*  sampleIndexes.push_back(i);*/
  /*}*/

  /*sort(sampleIndexes.begin(), sampleIndexes.end(),*/
  /*     [&sampleFreq](const int &a, const int &b) -> bool {*/
  /*       if (sampleFreq[a] != sampleFreq[b]) {*/
  /*         return sampleFreq[a] < sampleFreq[b];*/
  /*       } else {*/
  /*         return a < b;*/
  /*       }*/
  /*     });*/
}

void IVHD::time_step_R(bool firstStep) {
  if (firstStep) {
    initializeHelperVectors();
  } else {
    calcPositions<<<positions.size() / 256 + 1, 256>>>(positions.size(), d_samples);
  }

  // calculate forces
  calcForceComponents<<<distances.size() / 256 + 1, 256>>>(
      distances.size(), d_distances, d_samples);

  // calculate index of every force that should be applied for given sample
  applyForces<<<positions.size() / 256 + 1, 256>>>(
      positions.size(), d_samples);
}

bool IVHD::allocateInitializeDeviceMemory() {
  cuCall(hipMalloc(&d_positions, positions.size() * sizeof(float2)));
  cuCall(hipMalloc(&d_samples, positions.size() * sizeof(Sample)));
  cuCall(hipMalloc(&d_distances, distances.size() * sizeof(DistElem)));

  cuCall(hipMemcpy(d_positions, &positions[0],
                    sizeof(float2) * positions.size(), hipMemcpyHostToDevice));
  cuCall(hipMemset(d_samples, 0, positions.size() * sizeof(Sample)));
  cuCall(hipMemcpy(d_distances, &distances[0],
                    sizeof(DistElem) * distances.size(),
                    hipMemcpyHostToDevice));

  return true;
}

__global__ void copyPosRelease(int N, Sample *samples, float2 *positions) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i<N) {
    positions[i] = samples[i].pos;
    free(samples[i].components);
  }
}

bool IVHD::copyResultsToHost() {
  copyPosRelease<<<positions.size() / 256 + 1, 256>>>(positions.size(), d_samples, d_positions);
  cuCall(hipMemcpy(&positions[0], d_positions,
                    sizeof(float2) * positions.size(), hipMemcpyDeviceToHost));
  cuCall(hipFree(d_positions));
  cuCall(hipFree(d_distances));
  cuCall(hipFree(d_samples));

  return true;
}
