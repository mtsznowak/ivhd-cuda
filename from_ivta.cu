#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <algorithm>
#include <cassert>
#include <cstring>
#include <iostream>
#include <unordered_map>
#include "constants.h"
#include "from_ivta.h"
using namespace std;

anyVector3d IVHD::force(DistElem distance) {
  anyVector3d rv = positions[distance.i] - positions[distance.j];

  Real r = positions[distance.i].distance3D(positions[distance.j]);
  Real D = distance.r;

  Real energy = (r - D) / r;

  return rv * (-energy);
}

__global__ void calcPositions(long n, anyVector3d *v, anyVector3d *f,
                              anyVector3d *positions) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    v[i].x = v[i].x * a_factor + f[i].x * b_factor;
    v[i].y = v[i].y * a_factor + f[i].y * b_factor;
    positions[i].x += v[i].x;
    positions[i].y += v[i].y;
  }
  return;
}

__global__ void calcForceComponents(int compNumber, anyVector3d *components,
                                    DistElem *distances,
                                    anyVector3d *positions) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < compNumber) {
    DistElem distance = distances[i];
    anyVector3d posI = positions[distance.i];
    anyVector3d posJ = positions[distance.j];

    anyVector3d rv = posI;
    rv.x -= posJ.x;
    rv.y -= posJ.y;

    Real r = sqrtf((posI.x - posJ.x) * (posI.x - posJ.x) +
                   (posI.y - posJ.y) * (posI.y - posJ.y));
    Real D = distance.r;

    Real energy = (r - D) / r;
    rv.x *= -energy;
    rv.y *= -energy;

    // distances are sorted by their type
    if (distance.type == etRandom) {
      rv.x *= w_random;
      rv.y *= w_random;
    }
    components[i] = rv;
  }
  return;
}

__global__ void applyForces(int n, anyVector3d *f, DistElem *dstElems,
                            anyVector3d *components, int *lens,
                            int **dst_indexes, int *sample_indexes) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    i = sample_indexes[i];
    int dst_len = lens[i];

    for (int j = 0; j < dst_len; j++) {
      int dst_index = dst_indexes[i][j];
      DistElem &dst = dstElems[dst_index];
      int negat = ((i == dst.i) - (i == dst.j));
      anyVector3d &comp = components[dst_index];
      f[i].x += comp.x * negat;
      f[i].y += comp.y * negat;
    }
  }
  return;
}

void IVHD::initializeHelperVectors() {
  // calculate number of distances for each sample
  std::unordered_map<int, int> sampleFreq;
  for (const auto &dst : distances) {
    sampleFreq[dst.i]++;
    sampleFreq[dst.j]++;
  }

  // generate sorted (by the smallest number of distances) list of sample
  // indexes
  vector<int> sampleIndexes;
  for (int i = 0; i < positions.size(); i++) {
    sampleIndexes.push_back(i);
  }

  sort(sampleIndexes.begin(), sampleIndexes.end(),
       [&sampleFreq](const int &a, const int &b) -> bool {
         if (sampleFreq[a] != sampleFreq[b]) {
           return sampleFreq[a] < sampleFreq[b];
         } else {
           return a < b;
         }
       });

  cuCall(hipMalloc(&gpu_sample_indexes, sizeof(int) * positions.size()));
  cuCall(hipMemcpy(gpu_sample_indexes, &sampleIndexes[0],
                    sampleIndexes.size() * sizeof(int),
                    hipMemcpyHostToDevice));

  vector<vector<int>> dst_indexes_vec(positions.size());

  for (int i = 0; i < distances.size(); i++) {
    dst_indexes_vec[distances[i].i].push_back(i);
    dst_indexes_vec[distances[i].j].push_back(i);
  }

  cuCall(hipMalloc(&gpu_dst_indexes, positions.size() * sizeof(int *)));
  dst_indexes = (int **)malloc(positions.size() * sizeof(int *));

  for (int i = 0; i < positions.size(); i++) {
    cuCall(
        hipMalloc(&dst_indexes[i], dst_indexes_vec[i].size() * sizeof(int)));
    cuCall(hipMemcpy(dst_indexes[i], &dst_indexes_vec[i][0],
                      dst_indexes_vec[i].size() * sizeof(int),
                      hipMemcpyHostToDevice));
  }

  cuCall(hipMemcpy(gpu_dst_indexes, dst_indexes,
                    positions.size() * sizeof(int *), hipMemcpyHostToDevice));

  int sizes[positions.size()];
  for (int i = 0; i < positions.size(); i++) {
    sizes[i] = dst_indexes_vec[i].size();
  }
  cuCall(hipMalloc(&gpu_dst_lens, sizeof(int) * positions.size()));
  cuCall(hipMemcpy(gpu_dst_lens, sizes, sizeof(int) * positions.size(),
                    hipMemcpyHostToDevice));
}

void IVHD::time_step_R(bool firstStep) {
  if (firstStep) {
    hipMemset(gpu_v, 0, v.size() * sizeof(anyVector3d));
    initializeHelperVectors();
  } else {
    calcPositions<<<positions.size() / 256 + 1, 256>>>(positions.size(), gpu_v,
                                                       gpu_f, gpu_positions);
  }

  // calculate forces
  hipMemset(gpu_f, 0, f.size() * sizeof(anyVector3d));

  calcForceComponents<<<distances.size() / 256 + 1, 256>>>(
      distances.size(), gpu_components, gpu_distances, gpu_positions);

  // calculate index of every force that should be applied for given sample
  applyForces<<<positions.size() / 256 + 1, 256>>>(
      positions.size(), gpu_f, gpu_distances, gpu_components, gpu_dst_lens,
      gpu_dst_indexes, gpu_sample_indexes);
}

bool IVHD::allocateInitializeDeviceMemory() {
  cuCall(hipMalloc(&gpu_positions, positions.size() * sizeof(anyVector3d)));
  cuCall(hipMalloc(&gpu_v, v.size() * sizeof(anyVector3d)));
  cuCall(hipMalloc(&gpu_f, f.size() * sizeof(anyVector3d)));
  cuCall(hipMalloc(&gpu_distances, distances.size() * sizeof(DistElem)));
  cuCall(hipMalloc(&gpu_components, distances.size() * sizeof(anyVector3d)));

  cuCall(hipMemcpy(gpu_positions, &positions[0],
                    sizeof(anyVector3d) * positions.size(),
                    hipMemcpyHostToDevice));
  cuCall(hipMemcpy(gpu_v, &v[0], sizeof(anyVector3d) * v.size(),
                    hipMemcpyHostToDevice));
  cuCall(hipMemcpy(gpu_f, &f[0], sizeof(anyVector3d) * f.size(),
                    hipMemcpyHostToDevice));
  cuCall(hipMemcpy(gpu_distances, &distances[0],
                    sizeof(DistElem) * distances.size(),
                    hipMemcpyHostToDevice));

  return true;
}

bool IVHD::copyResultsToHost() {
  cuCall(hipMemcpy(&positions[0], gpu_positions,
                    sizeof(anyVector3d) * positions.size(),
                    hipMemcpyDeviceToHost));

  cuCall(hipFree(gpu_positions));
  cuCall(hipFree(gpu_v));
  cuCall(hipFree(gpu_f));
  cuCall(hipFree(gpu_distances));
  cuCall(hipFree(gpu_components));

  return true;
}
