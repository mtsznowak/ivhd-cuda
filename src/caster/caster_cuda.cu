#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <algorithm>
#include <cassert>
#include <cstring>
#include <iostream>
#include <unordered_map>
#include <thrust/execution_policy.h>
#include <thrust/device_ptr.h>
#include <thrust/reduce.h>
#include "constants.h"
#include "caster/caster_cuda.h"
using namespace std;

// initialize pos in Samples
// initialize num_components
__global__ void initializeSamples(int n, Sample *samples, float2 *positions,
    short *sampleFreq) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    Sample sample;
    sample.pos = positions[i];
    sample.v = {0, 0};
    sample.num_components = sampleFreq[i];
    // FIXME - malloc can return NULL
    sample.components =
      (float2 *)malloc(sample.num_components * sizeof(float2));
    samples[i] = sample;
  }
}

__global__ void initializeDistances(int nDst, DistElem *distances,
    short2 *dstIndexes, Sample *samples) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < nDst) {
    DistElem dst = distances[i];
    dst.comp1 = &samples[dst.i].components[dstIndexes[i].x];
    dst.comp2 = &samples[dst.j].components[dstIndexes[i].y];
    distances[i] = dst;
  }
}

void CasterCuda::initializeHelperVectors() {
  /*
   * calculate number of distances for each sample and index of each distance
   * for a given sample
   */
  short sampleFreq[positions.size()];
  for (int i = 0; i < positions.size(); i++) {
    sampleFreq[i] = 0;
  }

  short2 dstIndexes[distances.size()];

  for (int i = 0; i < distances.size(); i++) {
    dstIndexes[i] = {sampleFreq[distances[i].i]++,
      sampleFreq[distances[i].j]++};
  }

  // initialize samples
  short *d_sample_freq;
  cuCall(hipMalloc(&d_sample_freq, positions.size() * sizeof(short)));
  cuCall(hipMemcpy(d_sample_freq, sampleFreq, sizeof(short) * positions.size(),
        hipMemcpyHostToDevice));

  initializeSamples<<<positions.size() / 256 + 1, 256>>>(
      positions.size(), d_samples, d_positions, d_sample_freq);
  cuCall(hipFree(d_sample_freq));

  // initialize comps in Distances in device memory
  short2 *d_dst_indexes;
  cuCall(hipMalloc(&d_dst_indexes, distances.size() * sizeof(short2)));
  cuCall(hipMemcpy(d_dst_indexes, dstIndexes,
        sizeof(short2) * distances.size(), hipMemcpyHostToDevice));

  initializeDistances<<<distances.size() / 256 + 1, 256>>>(
      distances.size(), d_distances, d_dst_indexes, d_samples);
  cuCall(hipFree(d_dst_indexes));
}

/*
 * This function performs the preprocessing on the CPU that is optional
 *
 * Sorts samples by number of their distances and sorts distances by
 * index i or j to utilize cache better. After sorting samples, their indexes
 * change so we have to update distances once more
 */
void CasterCuda::sortHostSamples(vector<int> &labels) {
  // create array of sorted indexes
  vector<short> sampleFreq(positions.size());
  for (int i = 0; i < positions.size(); i++) {
    sampleFreq[i] = 0;
  }

  vector<int> sampleIndexes(positions.size());
  for (int i = 0; i < positions.size(); i++) {
    sampleIndexes[i] = i;
  }

  sort(sampleIndexes.begin(), sampleIndexes.end(),
      [&sampleFreq](const int &a, const int &b) -> bool {
      if (sampleFreq[a] != sampleFreq[b]) {
      return sampleFreq[a] < sampleFreq[b];
      } else {
      return a < b;
      }
      });

  // create mapping index->new index
  vector<int> newIndexes(positions.size());
  for (int i = 0; i < positions.size(); i++) {
    newIndexes[sampleIndexes[i]] = i;
  }

  // sort positions
  vector<float2> positionsCopy = positions;
  vector<int> labelsCopy = labels;
  for (int i = 0; i < positions.size(); i++) {
    positions[i] = positionsCopy[sampleIndexes[i]];
    labels[i] = labelsCopy[sampleIndexes[i]];
  }

  // update indexes in distances
  for (int i = 0; i < distances.size(); i++) {
    distances[i].i = newIndexes[distances[i].i];
    distances[i].j = newIndexes[distances[i].j];
  }

  // sort distances
  sort(distances.begin(), distances.end(),
      [](const DistElem &a, const DistElem &b) -> bool {
      if (a.i != b.i) {
      return a.i < b.i;
      } else {
      return a.j <= b.j;
      }
      });
}

bool CasterCuda::allocateInitializeDeviceMemory() {
  cuCall(hipMalloc(&d_positions, positions.size() * sizeof(float2)));
  cuCall(hipMalloc(&d_samples, positions.size() * sizeof(Sample)));
  cuCall(hipMalloc(&d_distances, distances.size() * sizeof(DistElem)));
  cuCall(hipMalloc(&d_errors, distances.size() * sizeof(float)));

  cuCall(hipMemcpy(d_positions, &positions[0],
        sizeof(float2) * positions.size(), hipMemcpyHostToDevice));
  cuCall(hipMemset(d_samples, 0, positions.size() * sizeof(Sample)));
  cuCall(hipMemset(d_errors, 0, distances.size() * sizeof(float)));
  cuCall(hipMemcpy(d_distances, &distances[0],
        sizeof(DistElem) * distances.size(),
        hipMemcpyHostToDevice));

  return true;
}

__global__ void copyPosRelease(int N, Sample *samples, float2 *positions) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N) {
    positions[i] = samples[i].pos;
    free(samples[i].components);
  }
}

void CasterCuda::prepare(vector<int> &labels){
  sortHostSamples(labels);
  allocateInitializeDeviceMemory();
}

void CasterCuda::finish(){
  copyResultsToHost();
}

__global__ void copyDevicePos(int N, Sample *samples, float2 *positions) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N) {
    positions[i] = samples[i].pos;
  }
}

void CasterCuda::copyPositions() {
  copyDevicePos<<<positions.size() / 256 + 1, 256>>>(positions.size(),
      d_samples, d_positions);
  cuCall(hipMemcpy(&positions[0], d_positions,
        sizeof(float2) * positions.size(), hipMemcpyDeviceToHost));
}

bool CasterCuda::copyResultsToHost() {
  copyPosRelease<<<positions.size() / 256 + 1, 256>>>(positions.size(),
      d_samples, d_positions);
  cuCall(hipMemcpy(&positions[0], d_positions,
        sizeof(float2) * positions.size(), hipMemcpyDeviceToHost));
  cuCall(hipFree(d_positions));
  cuCall(hipFree(d_distances));
  cuCall(hipFree(d_samples));

  return true;
}

__global__ void calculateErrors(int dstNum, DistElem *distances, Sample *samples, float *errors) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < dstNum;
      i += blockDim.x * gridDim.x) {
    DistElem dist = distances[i];
    float d = dist.r;
    float2 iPos = samples[dist.i].pos;
    float2 jPos = samples[dist.j].pos;
    float2 ij = {iPos.x - jPos.x, jPos.y - jPos.y};
    errors[i] = abs(d - sqrt(ij.x * ij.x + ij.y * ij.y));
  }
}

float CasterCuda::getError() {
  calculateErrors<<<256, 256>>>(distances.size(), d_distances,
      d_samples, d_errors);

  thrust::device_ptr<float> err_ptr = thrust::device_pointer_cast(d_errors);
  return thrust::reduce(err_ptr, err_ptr + distances.size(), 0, thrust::plus<float>());
}

void CasterCuda::simul_step() {
  if (!it++) {
    initializeHelperVectors();
  }

  simul_step_cuda();

  if(it % 100 == 0) {
    onError(getError());
  }

  if((itToPosReady--) == 0) {
    onPositions(positions);
  }

  if(it % 2000 == 0) {
    copyPositions();
    itToPosReady = 5;
    hipDeviceSynchronize();
  }
};
