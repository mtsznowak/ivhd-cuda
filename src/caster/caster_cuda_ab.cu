#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "constants.h"
#include "caster/caster_cuda_ab.h"
using namespace std;

__global__ void calcPositions(long n, Sample *samples) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n;
       i += blockDim.x * gridDim.x) {
    Sample sample = samples[i];

    for (int j = 0; j < sample.num_components; j++) {
      sample.f.x += sample.components[j].x;
      sample.f.y += sample.components[j].y;
    }

    sample.v.x = sample.v.x * a_factor + sample.f.x * b_factor;
    sample.v.y = sample.v.y * a_factor + sample.f.y * b_factor;
    sample.f = {0, 0};
    sample.pos.x += sample.v.x;
    sample.pos.y += sample.v.y;
    samples[i] = sample;
  }
  return;
}

__global__ void calcForceComponents(int compNumber, DistElem *distances,
                                    Sample *samples) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < compNumber;
       i += blockDim.x * gridDim.x) {
    DistElem distance = distances[i];

    float2 posI = samples[distance.i].pos;
    float2 posJ = samples[distance.j].pos;

    float2 rv = posI;
    rv.x -= posJ.x;
    rv.y -= posJ.y;

    float r = sqrtf((posI.x - posJ.x) * (posI.x - posJ.x) +
                    (posI.y - posJ.y) * (posI.y - posJ.y));
    float D = distance.r;

    float energy = (r - D) / r;
    rv.x *= -energy;
    rv.y *= -energy;

    // distances are sorted by their type
    if (distance.type == etRandom) {
      rv.x *= w_random;
      rv.y *= w_random;
    }
    *distance.comp1 = rv;
    *distance.comp2 = {-rv.x, -rv.y};
  }
  return;
}

void CasterCudaAB::simul_step_cuda() {
  calcForceComponents<<<64, 96>>>(distances.size(), d_distances, d_samples);
  calcPositions<<<64, 96>>>(positions.size(), d_samples);
}
